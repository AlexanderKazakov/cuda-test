#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cmath>
#include <fstream>
#include <unistd.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements. Each thread adds k elements of vectors.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          const int numElements, const int k) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	for (int j = 0; j < k; j++) {
		int i = threadIndex * k + j;
		if (i < numElements) C[i] = A[i] + B[i];
	}
}

int main(int argc, char** argv) {
	int numElements = 0;
	int k = 1;
	int opt = 0;
	while ((opt = getopt(argc, argv, "n:k:")) != -1) {
		switch (opt) {
			case 'n': numElements = atoi(optarg);
				break;
			case 'k': k = atoi(optarg);
				break;
			case '?':
			{
				fprintf(stderr, "Usage: ./addVector -n numElementsToAdd",
				                " -k numElementsToAddInOneThread\n");
				exit(-1);
			}
		}
	}
	if ((numElements < 1) || (k < 1)) {
		fprintf(stderr, "Bad parameters!\n");
		exit(-1);
	}
	printf("Adding vectors of size %d ...\n", numElements);
	
	// Allocate the host input vectors
	size_t size = numElements * sizeof (float);
	float *h_A = (float *) malloc(size);
	float *h_B = (float *) malloc(size);
	float *h_C = (float *) malloc(size);
	if (h_A == NULL || h_B == NULL || h_C == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}
	// Initialize the host input vectors
	srand(time(0));
	for (int i = 0; i < numElements; ++i) {
		h_A[i] = rand() / (float) RAND_MAX;
		h_B[i] = rand() / (float) RAND_MAX;
	}

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	
	// Allocate the device input vectors
	float *d_A = NULL;
	err = hipMalloc((void **) &d_A, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	float *d_B = NULL;
	err = hipMalloc((void **) &d_B, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	float *d_C = NULL;
	err = hipMalloc((void **) &d_C, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory 
	// to the device input vectors in device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", 
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 512;
	int blocksPerGrid = ((numElements + k - 1) / k + threadsPerBlock - 1) / threadsPerBlock;

	// Cuda Events for measuring of execution time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("CUDA kernel launch with %d blocks of %d threads\n", 
	       blocksPerGrid, threadsPerBlock);

	hipEventRecord(start);
	vectorAdd <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements, k);
	hipEventRecord(stop);

	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Evaluating execution time
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	// Evaluating computational bandwidth in GFLOPs
	float bandwidth = numElements / milliseconds / 1e+6;

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector C from device to host",
		                "(error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Verify that the result vector is correct
	for (int i = numElements - 5; i < numElements; ++i) {
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}
	srand(time(0));
	for (int counter = 0; counter < 10; counter++) {
		int i = (int) ( (rand() / (float) RAND_MAX) * numElements );
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	// Free device global memory
	err = hipFree(d_A);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", 
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipFree(d_B);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", 
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipFree(d_C);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", 
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	// Reset the device and exit
	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", 
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Write performance data to file for performance plot
	std::ofstream performanceData;
	performanceData.open("performanceData.txt", std::ios::app);
	performanceData << log10(numElements) << "\t" << bandwidth << "\n";
	performanceData.close();

	printf("Done in %f milliseconds with computational performance %f GFLOPs\n",
	       milliseconds, bandwidth);
	return 0;
}

